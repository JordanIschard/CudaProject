#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp> 
#include <vector>


// Matrix de convolution 
//
//  0  0 -1  0  0
//  0 -1 -2 -1  0
// -1 -2 16 -2 -1
//  0 -1 -2 -1  0
//  0  0 -1  0  0
__global__ void laplacian_of_gaussian(unsigned const char* data_in, unsigned char* const data_out, size_t rows, size_t cols)
{
    // On récupère les coordonnées du pixel
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    auto result = 0;

    if( i >= 2 && i < (rows - 2) && j >= 2 && j < (cols - 2) )
    {
        // Tous les pixels que l'on multiplie par 16
        result = data_in[3 * (i * cols + j)] * 16

        // Tous les pixels que l'on multiplie par -2
        + ( data_in[3 * ((i-1) * cols + j)] + data_in[3 * ((i+1) * cols + j)] + data_in[3 * (i * cols + (j-1))] + data_in[3 * (i * cols + (j+1))] ) * -2

        // Tous les pixels que l'on multiplie par -1
        + ( data_in[3 * ((i-2) * cols + j)] + data_in[3 * ((i+2) * cols + j)] + data_in[3 * (i * cols + (j-2))] + data_in[3 * (i * cols + (j+2))] 
            + data_in[3 * ((i-1) * cols + (j-1))] + data_in[3 * ((i-1) * cols + (j+1))] + data_in[3 * ((i+1) * cols + (j-1))] + data_in[3 * ((i+1) * cols + (j+1))] ) * -1;

        result = result * result;
        result > 255*255 ? result = 255*255 : result;

        data_out[ i * cols + j ] = sqrt((float)result);
    }
}

int main(int argc, char** argv)
{
    printf("Number of argument : %d\n", argc);

    if(argc == 2){

        // Mesure de temps
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        std::cout << "Création du timer faite" << std::endl;

        // Récupère l'image
        cv::Mat image_in = cv::imread(argv[1], cv::IMREAD_UNCHANGED);
        // Récupère les informations des pixels
        auto data_in = image_in.data;
        auto rows = image_in.rows;
        auto cols = image_in.cols;

	
        std::cout << "rows = " << rows << " columns = " << cols << std::endl;

        // On crée les informations de sorties 
        std::vector<unsigned char> out(rows * cols); 
        // On crée l'image de sortie
        cv::Mat image_out(rows, cols, CV_8UC1, out.data());

        std::cout << "Image et données de sortie initialisées" << std::endl;

        // On copie l'image d'entrée sur le device
        unsigned char * image_in_device;
        // On crée une copie des informations de sortie sur le device
        unsigned char* data_out_device;

        hipMalloc(&image_in_device, rows * cols);
    
	    auto err1 = hipGetLastError();
        if(err1 != hipSuccess){
            std::cout << "You're fuck up " << hipGetErrorString(err1) << std::endl;
        }else{
	        std::cout << "kuvheuio" << std::endl;
	    }
        std::cout << "Image sur le device allouée" << std::endl;

        hipMalloc(&data_out_device, rows * cols);

        std::cout << "Données de sortie sur le device allouées" << std::endl;

        hipMemcpy(image_in_device, data_in,  rows * cols, hipMemcpyHostToDevice );
                                                                                    
        std::cout << "Image d'entrée mise sur le device" << std::endl;

        dim3 threads(32, 32 );
        dim3 blocks(( cols -1 ) / threads.x + 1 , ( rows - 1) / threads.y + 1);

        std::cout << "Nombre de threads = " << threads.x << "  " << threads.y << std::endl;
        std::cout << "Nombre de blocks = " << blocks.x << "  " << blocks.y << std::endl;

        // Lancement du timer
        hipEventRecord(start);

        std::cout << "Lancement du timer" << std::endl;

        // lancement du programme
        laplacian_of_gaussian<<< blocks , threads >>>(image_in_device, data_out_device, rows, cols);

        // On arrête le timer
        hipEventRecord(stop);

        std::cout << "Fin du timer" << std::endl;

        hipDeviceSynchronize();
        auto err = hipGetLastError();
        if( err != hipSuccess )
        {
            printf("Errors found :\n %s", hipGetErrorString(err));
        }

        // On copie les informations de sortie du device vers le host
        hipMemcpy(out.data(), data_out_device, rows * cols, hipMemcpyDeviceToHost );
        
        // On récupère le temps d'exécution
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time : %f",milliseconds);

        cv::imwrite( "outCuda.jpg", image_out);

        // On libère l'espace sur le device
        hipFree(image_in_device);
        hipFree(data_out_device);
    }

    return 0;
}
